#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "voxel.h"
#include <unsupported/Eigen/CXX11/Tensor>
#include <hiprand.h>
#include <hipDNN.h>
#include <atomic>


#define NUM_THREADS 256

extern "C"  void fast_iter(int level, Voxel_L &lrImage, Voxel_E &dirSet, Eigen::Tensor<float, 3> &ss,
	int ss_x, int ss_y, int ss_z, Voxel_E &predict, int type_id);

extern "C" void fast_iter2(int level, Voxel_L &lrImage, Eigen::Tensor<unsigned char, 3> &dirSet, Eigen::Tensor<float, 3> &ss,
	int ss_x, int ss_y, int ss_z, Voxel_E &predict,int type_id, double thred, double thred2);

extern "C" void calc_direction_id(std::vector<Eigen::Tensor<float, 3> >kernels, Voxel_L &lrImage, std::vector<Eigen::Tensor<float, 3> > &res,
	Eigen::Tensor<unsigned char, 3> &idx_res);

extern "C" void test_conv(std::vector<Eigen::Tensor<float, 3>> &oris, Eigen::Tensor<float, 3> &kernel,
	 std::vector<Eigen::Tensor<float, 3>> &res);

void calc_mean_std(unsigned short *lrImaged, int total_pixel_num, double mean, double std);

__global__ void initialize(float *src, float val, int num);

__global__ void initialize(unsigned short *src, unsigned short val, int num);

__global__ void conv_mult(float *ori, float *kernel, float *res, int dim_x, int dim_y, int dim_z, int kernel_dim, int total_pixel_num);

__global__ void conv_mult(unsigned short *ori, float *kernel, float *res, int dim_x, int dim_y, int dim_z, int kernel_dim, int total_pixel_num);

__global__ void
calc_feature_map(unsigned short *lrImage_d, unsigned char *dirset_d, float *ss_d, int dim_x, int dim_y, int dim_z, 
	unsigned char *predict, int total_pixel_num, int ss_x, int ss_y, int ss_z, int level, int type_id, double thred, double thred2);

__global__ void
calc_feature_map_with_meanstd(unsigned short *lrImage_d, unsigned char *dirset_d, float *ss_d, int dim_x, int dim_y, int dim_z,
	unsigned char *predict, int total_pixel_num, int ss_x, int ss_y, int ss_z, int level, int type_id, double thred, double thred2,
	double mean, double std);

__global__ void pad_img(unsigned short *ori, unsigned short *tar, int padding_x, int padding_y, int padding_z, 
	                    int total_pixel_num, int dim_x, int dim_y, int dim_z);

__global__ void obtain_ids(unsigned short *idx, float*res, int total_pixel_num, int kernel_num);

extern "C" void conv(Voxel_L &lrImage, std::vector<Eigen::Tensor<float, 3> > kernels, int dim_x, int dim_y, int dim_z, int kernel_dim, bool reverse_xyz, int padding,
	std::vector<Eigen::Tensor<float, 3> > &res, Eigen::Tensor<unsigned char, 3> &id_res);


__device__ void normalize(float *vec)
{
	double L = std::sqrt(vec[0] * vec[0] + vec[1] * vec[1] + vec[2] * vec[2]);
	vec[0] = (vec[0]) / (L + 0.000000001);
	vec[1] = (vec[1]) / (L + 0.000000001);
	vec[2] = (vec[2]) / (L + 0.000000001);
}


__device__ void Coordinates(float *dir, float *res)
{
	float x = std::abs(dir[0]);
	float y = std::abs(dir[1]);
	float z = std::abs(dir[2]);
	int id = 0;
	if (x > y)
	{
		id = 1;
		if (z < y)
			id = 2;
	}
	else
	{
		if (z < x)
			id = 2;
	}
	int id0 = (id + 1) % 3;
	int id1 = (id + 2) % 3;
	res[0] = dir[0];
	res[1] = dir[1];
	res[2] = dir[2];

	res[id0 + 3] = dir[id1];
	res[id1 + 3] = - dir[id0];

	res[6] = res[1] * res[2 + 3] - res[2] * res[1 + 3];
	res[7] = -res[0] * res[2 + 3] + res[2] * res[0 + 3];
	res[8] = res[0] * res[1 + 3] - res[1] * res[0 + 3];
	normalize(res);
	normalize(res + 3);
	normalize(res + 6);
}

__device__ void set_value(float* adr, float val, int num)
{
	for (int i = 0; i < num; ++i)
	{
		adr[i] = val;
	}
}


__device__ void ImgLineSample(int *center, float *direction, int Num, unsigned short *lrImage_d, float *Points,
	int nx, int ny, int nz)
{
	double local_min = 999990000.0;
	double local_max = -10000.0;
	for (int i = 0; i < 2 * Num + 1; ++i)
	{
		int valid = 0;
		float p[3] = {center[0] + (i - Num) * direction[0], 
		              center[1] + (i - Num) * direction[1],
		              center[2] + (i - Num) * direction[2]};
		if (p[0] >= 0 && p[0] < nz - 1 &&
			p[1] >= 0 && p[1] < ny - 1 &&
			p[2] >= 0 && p[2] < nx - 1)
		{
			valid = 1;
		}
		if (valid)
		{
			int coor[3] = {std::round(p[0]), std::round(p[1]), std::round(p[2])};
			int I = lrImage_d[coor[0] + coor[1] * nx + coor[2] * nx * ny];
			if (I < local_min)
				local_min = I;
			if (I > local_max)
				local_max = I;
			Points[4 * i] = I;
			Points[4 * i + 1] = p[0];
			Points[4 * i + 2] = p[1];
			Points[4 * i + 3] = p[2];
		}
		else
		{
			Points[4 * i] = lrImage_d[center[0] + center[1] * nx + center[2] * nx * ny];
		}
	}
	/*for (int i = 0; i < 2 * Num + 1; ++i)
	{
		Points[4 * i] = (Points[4 * i] - local_min) / (local_max - local_min) * 2000;
	}*/
}


__device__ void ImgLineSampleSet(int *idx, float *res, const int Num, unsigned short *lrImage_d, int nx, int ny, int nz,
	float *Point0, float *Point1, float *Point2)
{
	set_value(Point0, 0.0, 4 * (2 * 4 + 1));
	set_value(Point1, 0.0, 4 * (2 * 4 + 1));
	set_value(Point2, 0.0, 4 * (2 * 4 + 1));
	ImgLineSample(idx, res, Num, lrImage_d, Point0, nx, ny, nz);
	ImgLineSample(idx, res + 3, Num, lrImage_d, Point1, nx, ny, nz);
	ImgLineSample(idx, res + 6, Num, lrImage_d, Point2, nx, ny, nz);
}

__device__ int Lineregiongrowing(float *Point, int index, float thred, int Point_num)
{
	int NUM = 0;
	for (int i = 0; i < Point_num - index; ++i)
	{
		if (Point[(i + index) * 4] > thred)
			NUM = NUM + 1;
		else
			break;
	}
	int NUM2 = 0;
	for (int i = 0; i < index; ++i)
	{
		if (Point[(index - i - 1) * 4] > thred)
			NUM2 = NUM2 + 1;
		else
			break;
	}
	return NUM + NUM2;
}


__device__ int Lineregiongrowing(float *Point, int index, float thred, float up_thred, int Point_num)
{
	int NUM = 0;
	for (int i = 0; i < Point_num - index; ++i)
	{
		if (Point[(i + index) * 4] > thred 
			&& Point[(i + index) * 4] < up_thred)
			NUM = NUM + 1;
		else
			break;
	}
	int NUM2 = 0;
	for (int i = 0; i < index; ++i)
	{
		if (Point[(index - i - 1) * 4] > thred 
			&& Point[(index - i - 1) * 4] < up_thred)
			NUM2 = NUM2 + 1;
		else
			break;
	}
	return NUM + NUM2;
}


__device__ void LineregiongrowingSET_NEW(float *Point0, float *Point1, float *Point2, int Point_num, int Index, double thred, float up_thred,
	                                     int *res_num)
{
	int num0 = Lineregiongrowing(Point0, Index, thred, up_thred, Point_num);
	int num1 = Lineregiongrowing(Point1, Index, thred, up_thred, Point_num);
	int num2 = Lineregiongrowing(Point2, Index, thred, up_thred, Point_num);
	res_num[0] = num0;
	res_num[1] = num1;
	res_num[2] = num2;
}


__device__ void LineregiongrowingSET(float *Point0, float *Point1, float *Point2, int Point_num, int Index, double thred, int *res_num)
{
	int num0 = Lineregiongrowing(Point0, Index, thred, Point_num);
	int num1 = Lineregiongrowing(Point1, Index, thred, Point_num);
	int num2 = Lineregiongrowing(Point2, Index, thred, Point_num);
	res_num[0] = num0;
	res_num[1] = num1;
	res_num[2] = num2;
}


__global__ void conv_mult(float *ori, float *kernel, float *res, int dim_x, int dim_y, int dim_z, int kernel_dim, int total_pixel_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int padding = (kernel_dim - 1) / 2;
	if (idx < total_pixel_num)
	{
		int z = idx / (dim_x * dim_y);
		int xy = idx % (dim_x * dim_y);
		int y = xy / dim_x;
		int x = xy % dim_x;
		float value = 0.0;
		for (int i = 0; i < kernel_dim; ++i)   //z
		{
			for (int j = 0; j < kernel_dim; ++j)  //y
			{
				for (int k = 0; k < kernel_dim; ++k) //x
				{
					/*if (x == 0 && y == 5 && z == 5)
					{
						printf("%f, %f,%d, %d, %d, %d, %d, %d\n", ori[(x + k) + (y + j) * (dim_x + padding * 2) + (z + i) * (dim_x + padding * 2) * (dim_y + padding * 2)],
							kernel[k + j * kernel_dim + i * kernel_dim * kernel_dim], k, j, i, x + k, y + j, z + i);
					}*/
					value += ori[(x + k) + (y + j) * (dim_x + padding * 2) + (z + i) * (dim_x + padding * 2) * (dim_y + padding * 2)]
						* kernel[k + j * kernel_dim + i * kernel_dim * kernel_dim];
				}
			}
		}
		/*if (x == 0 && y == 5 && z == 5)
		{
			printf("%f\n", value);
		}*/
		res[idx] = value;
	}
}
//__global__ void conv_mult_new(unsigned short *ori, float *kernel, float *res, int dim_x, int dim_y, int dim_z, int kernel_dim, int total_pixel_num, int kernel_num)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	int padding = (kernel_dim - 1) / 2;
//	if (idx < total_pixel_num)
//	{
//		int z = idx / (dim_x * dim_y);
//		int xy = idx % (dim_x * dim_y);
//		int y = xy / dim_x;
//		int x = xy % dim_x;
//		float value = 0.0;
//		for (int i = 0; i < kernel_dim; ++i)   //z
//		{
//			for (int j = 0; j < kernel_dim; ++j)  //y
//			{
//				for (int k = 0; k < kernel_dim; ++k) //x
//				{
//					/*	value += ori[(x + k) + (y + j) * (dim_x + padding * 2) + (z + i) * (dim_x + padding * 2) * (dim_y + padding * 2)]
//							* kernel[k + j * kernel_dim + i * kernel_dim * kernel_dim];*/  // conv with xyz
//
//					value += ori[(z + i) + (y + j) * (dim_x + padding * 2) + (x + k) * (dim_x + padding * 2) * (dim_y + padding * 2)]
//						   * kernel[k + j * kernel_dim + i * kernel_dim * kernel_dim];
//				}
//			}
//		}
//		res[idx] = value;
//	}
//}


__global__ void conv_mult(unsigned short *ori, float *kernel, float *res, int dim_x, int dim_y, int dim_z, int kernel_dim, int total_pixel_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int padding = (kernel_dim - 1) / 2;
	if (idx < total_pixel_num)
	{
		int z = idx / (dim_x * dim_y);
		int xy = idx % (dim_x * dim_y);
		int y = xy / dim_x;
		int x = xy % dim_x;
		float value = 0.0;
		for (int i = 0; i < kernel_dim; ++i)   //z
		{
			for (int j = 0; j < kernel_dim; ++j)  //y
			{
				for (int k = 0; k < kernel_dim; ++k) //x
				{
					// conv with xyz
					/*value += ori[(x + k) + (y + j) * (dim_x + padding * 2) + (z + i) * (dim_x + padding * 2) * (dim_y + padding * 2)]
						* kernel[k + j * kernel_dim + i * kernel_dim * kernel_dim];  */

					value += ori[(z + i) + (y + j) * (dim_x + padding * 2) + (x + k) * (dim_x + padding * 2) * (dim_y + padding * 2)]
						* kernel[k + j * kernel_dim + i * kernel_dim * kernel_dim];
				}
			}
		}
		/*if (x == 0 && y == 5 && z == 5)
		{
			printf("%f\n", value);
		}*/
		res[idx] = value;
	}
}


__global__ void pad_img(float *ori, float *tar, int padding_x, int padding_y, int padding_z,
	int total_pixel_num, int dim_x, int dim_y, int dim_z)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		int z = idx / (dim_x * dim_y);
		int xy = idx % (dim_x * dim_y);
		int y = xy / dim_x;
		int x = xy % dim_x;
		tar[x + padding_x + (y + padding_y) * (dim_x + padding_x * 2) + (z + padding_z) * (dim_x + padding_x * 2) * (dim_y + padding_y * 2)] = ori[idx];
	}
}


__global__ void obtain_ids(unsigned char *res, float *src, int total_pixel_num, int kernel_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		int res_id = 0;
		float value = -1000.0;
		for (int i = 0; i < kernel_num; ++i)
		{
			float tmp_res = src[i * total_pixel_num + idx];
			if (tmp_res > value)
			{
				value = tmp_res;
				res_id = i;
			}
		}
		res[idx] = res_id;
	}
}


__global__ void pad_img(unsigned short *ori, unsigned short *tar, int padding_x, int padding_y, int padding_z,
	int total_pixel_num, int dim_x, int dim_y, int dim_z)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		int z = idx / (dim_x * dim_y);
		int xy = idx % (dim_x * dim_y);
		int y = xy / dim_x;
		int x = xy % dim_x;
		tar[x + padding_x + (y + padding_y) * (dim_x + padding_x * 2) + (z + padding_z) * (dim_x + padding_x * 2) * (dim_y + padding_y * 2)] = ori[idx];
	}
}


__global__ void
Inttofloat(unsigned short *lrImage_d, float *tar, int total_pixel_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		tar[idx] = lrImage_d[idx];
	}
}


__device__ bool calc_ratio(int *center, unsigned short *lrImage_d, int dim_x, int dim_y, int dim_z)
{
	double dif = -1000.0;
	double I = lrImage_d[center[0] + center[1] * dim_x + center[2] * dim_x * dim_y];
	bool negative = false;
	for (int x = 0; x < 3; ++x)
	{
		for (int y = 0; y < 3; ++y)
		{
			for (int z = 0; z < 3; ++z)
			{
				double temp_dif = I
					- lrImage_d[center[0] + x - 1 + (center[1] + y - 1) * dim_x + (center[2] + z - 1) * dim_x * dim_y];
				if (std::abs(temp_dif) > dif)
				{
					dif = temp_dif;
					if (temp_dif < 0)
						negative = true;
					else
						negative = false;
				}
			}
		}
	}
	return negative;
	
	/*if (double(dif) / I < 1.0 && !negative)
		return (1 - double(dif) / I) * 0.5;
	
	return 1.0;*/
}


__global__ void calc_valid_number(unsigned short *lrImage_d, int total_pixel_num, int *valid)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num && lrImage_d[idx] > 0.0001)
	{
		atomicAdd(valid, 1);
	}
}


__global__ void calc_mean(unsigned short *lrImage_d, int total_pixel_num, double *sum, int nonzero)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		double mean = float(lrImage_d[idx]) / nonzero;
		atomicAdd(sum, mean);
	}
}


__global__ void calc_std(unsigned short *lrImage_d, int total_pixel_num, double mean, double *sum, int nonzero)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		float square = (lrImage_d[idx] - mean) * (lrImage_d[idx] - mean) / nonzero;
		atomicAdd(sum, square);
	}
}


__global__ void calc_std(unsigned short *lrImage_d, int total_pixel_num, float mean, float *sum)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		float square = (lrImage_d[idx] - mean) * (lrImage_d[idx] - mean) / total_pixel_num;
		atomicAdd(sum, square);
	}
}


__global__ void
calc_feature_map(unsigned short *lrImage_d, unsigned char *dirset_d, float *ss_d, int dim_x, int dim_y, int dim_z, 
	unsigned char *predict, int total_pixel_num, int ss_x, int ss_y, int ss_z, int level, int type_id, double thred, double thred2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		int i = idx / (dim_x * dim_y);    // z
		int ii = idx % (dim_x * dim_y);  
		int j = ii / dim_x;              // y
		int k = ii % dim_x;              // x
		/*int id = dirset_d[k + dim_x * j + ii];*/
		int id = dirset_d[idx];
		float dirx[3] = { ss_d[id + 2 * ss_x] - ss_d[id],
						  ss_d[id + 2 * ss_x + 1 * ss_x * ss_y] - ss_d[id + 1 * ss_x * ss_y],
						  ss_d[id + 2 * ss_x + 2 * ss_x * ss_y] - ss_d[id + 2 * ss_x * ss_y] };
		float res[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
		Coordinates(dirx, res);
		float *Point_v;
		int center[3] = { k, j, i };
		float Point0[4 * (2 * 4 + 1)], Point1[4 * (2 * 4 + 1)], Point2[4 * (2 * 4 + 1)];
		ImgLineSampleSet(center, res, 4, lrImage_d, dim_x, dim_y, dim_z, Point0, Point1, Point2);
		double ratio = 1.0;
		bool negative = true;
		if(k > 0 && k < dim_x - 1 && j > 0 && j < dim_y - 1 && i > 0 && i < dim_z - 1)
			negative = calc_ratio(center, lrImage_d, dim_x, dim_y, dim_z);
		ratio = 0.2;

		float thres = Point0[4 * 4];

		if (thres != 0 )
		{
			double a = 0.1 * (7 - level) * thres * ratio; // 7 - lve
			//double b = (7 - level) * 0.3;
			double thre_sub = a;
			//if (a > b)
				//thre_sub = a;
			if ((thres - thre_sub - 1) > thred)  // 20  lightsheet 10.0
			{
				int res_num[3] = { 0, 0, 0 };
				/*LineregiongrowingSET(Point0, Point1, Point2, 4 * (2 * 4 + 1), 4, thres - thre_sub - 1, res_num);*/
				if (type_id > 2)
					LineregiongrowingSET_NEW(Point0, Point1, Point2, 4 * (2 * 4 + 1), 4, thres - thre_sub - 1, thres + thre_sub, res_num);
				else
					LineregiongrowingSET_NEW(Point0, Point1, Point2, 4 * (2 * 4 + 1), 4, thres - thre_sub - 1, 70000, res_num);
				double p = 0.0;
				if ((type_id % 3) == 0)
				{
					p = double(res_num[0]) / 9.0;
				}
				else if ((type_id % 3) == 1)
				{
					p = double(res_num[1] * res_num[2]) / (9 * 9);
				}
				else if ((type_id % 3) == 2)
				{
					/*p = double(res_num[0] * res_num[1] * res_num[2]) / (9 * 9 * 9);*/
					double num = res_num[0];
					if (num < res_num[1])
					{
						num = res_num[1];
					}
					if (num < res_num[2])
						num = res_num[2];
					p = double(num) / 9;
				}

				predict[idx] = int((1 - p) * 255);
			}
			else
			{
				predict[idx] = 0.0;
			}
		}
		/*else
		{
			if (level < 3)
			{
				predict[idx] = 0.0;
			}
			else
			{
				double p = double(1 * 1) / (9 * 9);
				predict[idx] = int((1 - p) * 255);
			}
		}*/
	}
}

__global__ void
calc_feature_map_with_meanstd(unsigned short *lrImage_d, unsigned char *dirset_d, float *ss_d, int dim_x, int dim_y, int dim_z,
	unsigned char *predict, int total_pixel_num, int ss_x, int ss_y, int ss_z, int level, int type_id, double thred, double thred2,
	double mean, double std)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_pixel_num)
	{
		int i = idx / (dim_x * dim_y);    // z
		int ii = idx % (dim_x * dim_y);
		int j = ii / dim_x;              // y
		int k = ii % dim_x;              // x
		/*int id = dirset_d[k + dim_x * j + ii];*/
		int id = dirset_d[idx];
		float dirx[3] = { ss_d[id + 2 * ss_x] - ss_d[id],
						  ss_d[id + 2 * ss_x + 1 * ss_x * ss_y] - ss_d[id + 1 * ss_x * ss_y],
						  ss_d[id + 2 * ss_x + 2 * ss_x * ss_y] - ss_d[id + 2 * ss_x * ss_y] };
		float res[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
		Coordinates(dirx, res);
		float *Point_v;
		int center[3] = { k, j, i };
		float Point0[4 * (2 * 4 + 1)], Point1[4 * (2 * 4 + 1)], Point2[4 * (2 * 4 + 1)];
		ImgLineSampleSet(center, res, 4, lrImage_d, dim_x, dim_y, dim_z, Point0, Point1, Point2);
		double ratio = 1.0;
		bool negative = true;
		/*if (k > 0 && k < dim_x - 1 && j > 0 && j < dim_y - 1 && i > 0 && i < dim_z - 1)
			negative = calc_ratio(center, lrImage_d, dim_x, dim_y, dim_z);*/
		
		float thres = Point0[4 * 4];
		//if (thres > mean  + 1.1 * std -  std * 0.15 * (level + 1))
	   if (1/*thres > mean + 1.1 * std - std * 0.15 * (level + 1)*/)
		{
			//double a = 0.1 * (7 - level * 0.5) * std; // 7 - lve
			double a = thred2 * (1 + level) * std; // 7 - lve // 0.4, 0.2, 0.1
			/*if (thres < mean + std)
				a = 0.2 * (1 + level) * std;*/
			//a = 0.1 * (7 - level) * thres;
			//double b = (7 - level) * 0.3;
			double thre_sub = a;
			//if (a > b)
				//thre_sub = a;
			if (1/*(thres - thre_sub - 1) > thred*/)  // 20  lightsheet 10.0
			{
				int res_num[3] = { 0, 0, 0 };
				/*LineregiongrowingSET(Point0, Point1, Point2, 4 * (2 * 4 + 1), 4, thres - thre_sub - 1, res_num);*/
				if (type_id > 2)
					LineregiongrowingSET_NEW(Point0, Point1, Point2, 4 * (2 * 4 + 1), 4, thres - thre_sub - 1, thres + thre_sub, res_num);
				else
					LineregiongrowingSET_NEW(Point0, Point1, Point2, 4 * (2 * 4 + 1), 4, thres - thre_sub - 1, 70000, res_num);
				double p = 0.0;
				if ((type_id % 3) == 0)
				{
					p = double(res_num[0]) / 9.0;
				}
				else if ((type_id % 3) == 1)
				{
					p = double(res_num[1] * res_num[2]) / (9 * 9);
				}
				else if ((type_id % 3) == 2)
				{
					/*p = double(res_num[0] * res_num[1] * res_num[2]) / (9 * 9 * 9);*/
					double num = res_num[0];
					if (num < res_num[1])
					{
						num = res_num[1];
					}
					if (num < res_num[2])
						num = res_num[2];
					p = double(num) / 9;
				}

				predict[idx] = int((1 - p) * 255);
			}
			else
			{
				predict[idx] = 0.0;
			}
		}
		/*else
		{
			if (level < 3)
			{
				predict[idx] = 0.0;
			}
			else
			{
				double p = double(1 * 1) / (9 * 9);
				predict[idx] = int((1 - p) * 255);
			}
		}*/
	}
}

void compute_stride(const int * size, int*stride)
{
	for (int i = 4; i >= 0; i--)
		stride[i] = (i == 4) ? 1 : size[i + 1] * stride[i + 1];
}


extern "C" void conv_new(Voxel_L &lrImage, std::vector<Eigen::Tensor<float, 3> > kernels, int dim_x, int dim_y, int dim_z, int kernel_dim, bool reverse_xyz, int padding,
	std::vector<Eigen::Tensor<float, 3> > &res_C, Eigen::Tensor<unsigned char, 3> &id_res_c)
{
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
	int total_pixel_num = lrImage.dim_x * lrImage.dim_y * lrImage.dim_z;
	unsigned short* lrImage_d;
	hipMalloc(&lrImage_d, sizeof(unsigned short) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z);
	for (int i = 0; i < lrImage.dim_z; ++i)
	{
		hipMemcpy(lrImage_d + i * lrImage.dim_x * lrImage.dim_y, lrImage.data_ptr[i], sizeof(unsigned short) *
			lrImage.dim_x * lrImage.dim_y, hipMemcpyHostToDevice);
	}
	float *f_lrImage_d;
	hipMalloc(&f_lrImage_d, sizeof(float) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z);
	Inttofloat << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (lrImage_d, f_lrImage_d, total_pixel_num);

	float *d_output;
	
	// 0 ����cudnn���
	hipdnnHandle_t cudnn;
	auto cudnnHandle = hipdnnCreate(&cudnn);
	if (cudnnHandle != HIPDNN_STATUS_SUCCESS) {
		std::cout << "����cudnn�����ʧ�ܣ�" << std::endl;

	}

	//1 �������ݺͼ������������
	// 1.1 ������������������
	int q = 1, r = 1, m = dim_x, n = dim_y, p = dim_z;
	//���ﴴ��5ά�����ԭ���Ǹ�ά�Ⱦ������ʱ���ٷ��ĵ��Ƽ�ʹ�� >= 4ά���������м��㣬����Ҫ��ά�ȶ���Ϊ1����
	int inputDims[5] = { q,r,m, n, p }; // ���������ĳߴ� 
	int input_stride[5]; //���������������Ĳ���------**��Ҫ**��Ҫ**��Ҫ**
	compute_stride(inputDims, input_stride);
	hipdnnTensorDescriptor_t inputDesc;//��������������
	hipdnnCreateTensorDescriptor(&inputDesc);
	hipdnnStatus_t status = hipdnnSetTensorNdDescriptor(inputDesc, HIPDNN_DATA_FLOAT, 5, inputDims, input_stride);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cout << "��������������������ʧ�ܣ�" << std::endl;

	}

	int kernel_num = 108;
	// 1.2 ���������������
	int filterDims[5] = { kernel_num, 1, 13, 13, 13 };   // ����˳ߴ�
	hipdnnFilterDescriptor_t filterDesc;
	hipdnnCreateFilterDescriptor(&filterDesc);
	status = hipdnnSetFilterNdDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 5, filterDims);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cout << "�����������������ʧ�ܣ�" << std::endl;

	}

	// 1.3 ��������������������
	int  conmv_padA[3] = { 6, 6, 6 };//��䣬��ʾ�ظ���ά�Ȳ�0����������Ϊ�˽����������ݳߴ���С�����⣬��Ϊȫ0���ʾ����Ҫ���
	int conv_filterStrideA[3] = { 1,1,1 };//���ʱʹ�þ���˵Ĳ�����ȫ1��ʾ����������Ϊ��������
	int conv_dilationA[3] = { 1,1,1 };//arrayLength��arg 2)������ָʾ��ÿ��ά����������,����������ǶԾ���˲����ģ�ĳ��ά������ϵ��>1ʱ����Ѿ�������ά�ȷŴ��м��ȱʧ������0���룻ȫ1��ʾ������
	hipdnnConvolutionDescriptor_t convDesc;
	hipdnnCreateConvolutionDescriptor(&convDesc);
	status = hipdnnSetConvolutionNdDescriptor(convDesc, 3, conmv_padA, conv_filterStrideA, conv_dilationA, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cout << "�������������������ʧ�ܣ�" << std::endl;

	}

	// 1.4 ������������ĳߴ�,�Լ�����Ҳ���ԣ����ǻ��ǽ��������ĺ��������㣬�պÿ�����֤֮ǰ���������������Ƿ������Լ�Ԥ��
	// outputDim = 1 + ( inputDim + 2.*pad - (((filterDim-1).*dilation)+1) )./convolutionStride
	int outputDims[5];
	status = cudnnGetConvolutionNdForwardOutputDim(convDesc, inputDesc, filterDesc, 5, outputDims);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cout << "������������ĳߴ磺ʧ�ܣ�" << std::endl;

	}
	
	/*std::cout << "Output size: ";
	for (int i = 0; i < sizeof(outputDims) / sizeof(float); i++) {
		if (i < sizeof(outputDims) / sizeof(float) - 1) {
			std::cout << outputDims[i];
			std::cout << " X ";
		}
	}
	std::cout << outputDims[sizeof(outputDims) / sizeof(float) - 1] << std::endl;*/

	// 1.5 �����������������
	hipdnnTensorDescriptor_t outputDesc;
	hipdnnCreateTensorDescriptor(&outputDesc);
	int output_stride[5]; //��������������Ĳ���
	compute_stride(outputDims, output_stride);
	status = hipdnnSetTensorNdDescriptor(outputDesc, HIPDNN_DATA_FLOAT, 5, outputDims, output_stride);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cout << "�������������������ʧ�ܣ�" << std::endl;

	}

	// 2 ���ݺͼ����ڴ�ռ�������ʼ��
	//2.1 ��������������ڴ��С
	size_t in_bytes = 0;//�������������ڴ�
	status = cudnnGetTensorSizeInBytes(inputDesc, &in_bytes);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "fail to get bytes of in tensor: " << hipdnnGetErrorString(status) << std::endl;

	}
	size_t  out_bytes = 0;//������������ڴ�
	status = cudnnGetTensorSizeInBytes(outputDesc, &out_bytes);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "fail to get bytes of out tensor: " << hipdnnGetErrorString(status) << std::endl;

	}
	size_t filt_bytes = 1;//����������ڴ�
	for (int i = 0; i < sizeof(filterDims) / sizeof(int); i++) {
		filt_bytes *= filterDims[i];
	}
	filt_bytes *= sizeof(float);
	//�Զ�Ѱ�����ž�����㷽��,�����Զ�ѡ��������㷨������perfResults�ṹ����
	int returnedAlgoCount = 0;
	hipdnnConvolutionFwdAlgoPerf_t perfResults;
	status = cudnnGetConvolutionForwardAlgorithm_v7(cudnn, inputDesc, filterDesc, convDesc, outputDesc, 1, &returnedAlgoCount, &perfResults);
	if (returnedAlgoCount != 1 || status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "�Զ����������㷽����ʧ�ܣ�" << std::endl;

	}
	//����������������ڴ�ռ��С��������workspace_bytes������
	size_t workspace_bytes{ 0 };
	status = hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, perfResults.algo, &workspace_bytes);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cout << "����������������ڴ�ռ䣺ʧ�ܣ�" << std::endl;

	}

	//2.2 �ж�GPU���Ƿ����㹻���ڴ�ռ����ڼ���
	size_t request = in_bytes + out_bytes + workspace_bytes + filt_bytes;
	size_t cudaMem_free = 0, cudaMem_total = 0;
	hipError_t cuda_err = hipMemGetInfo(&cudaMem_free, &cudaMem_total);
	if (cuda_err != hipSuccess) {
		std::cerr << "fail to get mem info: " << hipGetErrorString(cuda_err) << std::endl;

	}
	if (request > cudaMem_free) {
		std::cout << in_bytes / (1024.0 * 1024.0 * 1024.0) << " " << out_bytes / (1024.0 * 1024.0 * 1024.0) << " "
			<< workspace_bytes / (1024.0 * 1024.0 * 1024.0) << " " << filt_bytes / (1024.0 * 1024.0 * 1024.0) << std::endl;
		std::cerr << request / (1024.0 * 1024.0 * 1024.0) << " " << cudaMem_free / (1024.0 * 1024.0 * 1024.0) << " not enough gpu memory to run" << std::endl;

	}

	//2.2 �������Ϸ����ڴ�洢��������������˺��������


	// 2.3 ���ݳ�ʼ��
	// cudaҪ��������һά���ݣ�NCHW��ʽ��,���Խ�ԭ����reshape��һ������,���뵽GPU֮�������������е�ά�Ȳ�����ԭ�ģ����Բ��õ���,
	// ���۶���ά�����ݣ� ��˳��reshapeΪһ�м���(����NCHW��ʽ��NHWC��ʽ��reshape��ʽ�ǲ�һ���ģ�һ��Ҫע�⣬������ƥ�伴�ɣ�
	// 2.3.1 ��������   

	//2.3.2 ��������ݣ������ٵ����ӣ����Ը����Լ��������Լ��ģ�ֻҪά�Ⱥ�ǰ�涨��ľ����ά��һ�¼���


	// 2.4 ���豸��gpu)�Ϸ����ڴ�ռ�
	hipMalloc(&d_output, out_bytes);
	void* d_workspace{ nullptr };
	hipMalloc(&d_workspace, workspace_bytes);

	// 3 �����������;���˿������豸

	float* kernel_d;
	hipMalloc(&kernel_d, sizeof(float) * kernel_dim * kernel_dim * kernel_dim * kernel_num);
	for (int i = 0; i < kernel_num; ++i)
		hipMemcpy(kernel_d + i * kernel_dim * kernel_dim * kernel_dim, kernels[i].data(), sizeof(float) * kernel_dim * kernel_dim * kernel_dim, hipMemcpyHostToDevice);


	// 4 ���о������
	float alpha = 1.0f, beta = 0.0f;
	status = hipdnnConvolutionForward(cudnn, &alpha, inputDesc, f_lrImage_d, filterDesc, kernel_d, convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, d_workspace, workspace_bytes, &beta, outputDesc, d_output);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cout << "���������̣�ʧ�ܣ�" << std::endl;

	}
	hipEventRecord(end);
	hipEventSynchronize(end);

	// 5 ��������󿽱�������
	//Eigen::Tensor<float, 3> res_c(128, 128, 128);
	/*for (int i = 0; i < 108; ++i)
	{
		hipMemcpy(res_c.data(), d_output + i * 128 * 128 * 128, sizeof(float) * 128 * 128 * 128, hipMemcpyDeviceToHost);
		res_C.push_back(res_c);
	}*/
	//��ӡ�������
	/*for (int i = 0; i < out_bytes; i++) {
		std::cout << output[i] << " ";
	}*/


	// 6 �ͷ���Դ
	//6.1 �ͷ���������ռ�õ��ڴ�

	
	//6.2 �ͷ�������ռ�õ��ڴ�
	status = hipdnnDestroyTensorDescriptor(inputDesc);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "fail to destroy input tensor desc: " << hipdnnGetErrorString(status) << std::endl;

	}
	status = hipdnnDestroyFilterDescriptor(filterDesc);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "fail to destroy filter tensor desc: " << hipdnnGetErrorString(status) << std::endl;

	}
	status = hipdnnDestroyConvolutionDescriptor(convDesc);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "fail to destroy conv desc: " << hipdnnGetErrorString(status) << std::endl;

	}
	status = hipdnnDestroyTensorDescriptor(outputDesc);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "fail to destroy outout tensor desc: " << hipdnnGetErrorString(status) << std::endl;

	}
	//6.3 �ͷ�cudnn����ڴ�
	status = hipdnnDestroy(cudnn);
	if (status != HIPDNN_STATUS_SUCCESS) {
		std::cerr << "fail to destroy cudnn handle" << std::endl;

	}
	//6.4 �ͷ������ϴ洢������ڴ�
	std::cout << "Done." << std::endl;

	float time;
	hipEventElapsedTime(&time, start, end);
	std::cout << "GPU Time: " << time << std::endl;
	
	
	/*for (int i = 0; i < kernels.size(); ++i)
	{
		float* kernel_d;
		hipMalloc(&kernel_d, sizeof(float) * kernel_dim * kernel_dim * kernel_dim);
		hipMemcpy(kernel_d, kernels[i].data(), sizeof(float) * kernel_dim * kernel_dim * kernel_dim, hipMemcpyHostToDevice);

		Eigen::Tensor<float, 3> res_c(128, 128, 128);
		conv_mult << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (padded_img_d, kernel_d, res + i * total_pixel_num,
			lrImage.dim_x, lrImage.dim_y, lrImage.dim_z, kernel_dim, total_pixel_num);
		hipMemcpy(res_c.data(), res + i * total_pixel_num, sizeof(float) * 128 * 128 * 128, hipMemcpyDeviceToHost);
		res_C.push_back(res_c);
	}*/

	unsigned char *id_res;
	hipMalloc(&id_res, sizeof(unsigned char*) * total_pixel_num);
	obtain_ids << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (id_res, d_output, total_pixel_num, kernels.size());
	hipMemcpy(id_res_c.data(), id_res, sizeof(unsigned char) * total_pixel_num, hipMemcpyDeviceToHost);

	// free memory
	{
		hipFree(lrImage_d);
		hipFree(f_lrImage_d);
		hipFree(d_output);
		hipFree(d_workspace);
		hipFree(kernel_d);
		hipFree(id_res); 
		cuda_err = hipFree(d_output);
		
	}
}


extern "C" void conv(Voxel_L &lrImage, std::vector<Eigen::Tensor<float, 3> > kernels, int dim_x, int dim_y, int dim_z, int kernel_dim, bool reverse_xyz, int padding,
	std::vector<Eigen::Tensor<float, 3> > &res_C, Eigen::Tensor<unsigned char, 3> &id_res_c)
{	
	unsigned short* lrImage_d;
	hipMalloc(&lrImage_d, sizeof(unsigned short) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z);
	for (int i = 0; i < lrImage.dim_z; ++i)
	{
		hipMemcpy(lrImage_d + i * lrImage.dim_x * lrImage.dim_y, lrImage.data_ptr[i], sizeof(unsigned short) *
			lrImage.dim_x * lrImage.dim_y, hipMemcpyHostToDevice);
	}
	unsigned short* padded_img_d;
	hipMalloc(&padded_img_d, sizeof(unsigned short) * (lrImage.dim_x + padding * 2)
											* (lrImage.dim_y + padding * 2)
											* (lrImage.dim_z + padding * 2));
	int total_padded_num = (lrImage.dim_x + padding * 2) * (lrImage.dim_y + padding * 2) * (lrImage.dim_z + padding * 2);
	unsigned short initial_val = 0;
	initialize <<< (total_padded_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (padded_img_d, initial_val, total_padded_num);

	int total_pixel_num = lrImage.dim_x * lrImage.dim_y * lrImage.dim_z;
	pad_img << <(total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (lrImage_d, 
		padded_img_d, padding, padding, padding, total_pixel_num, lrImage.dim_x, lrImage.dim_y, lrImage.dim_z);

	float *res;
	hipMalloc(&res, sizeof(float) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z * kernels.size());

	// original version.
	//for (int i = 0; i < kernels.size(); ++i)
	//{
	//	float* kernel_d;
	//	hipMalloc(&kernel_d, sizeof(float) * kernel_dim * kernel_dim * kernel_dim);
	//	hipMemcpy(kernel_d, kernels[i].data(), sizeof(float) * kernel_dim * kernel_dim * kernel_dim, hipMemcpyHostToDevice);

	//	Eigen::Tensor<float, 3> res_c(128, 128, 128);
	//	conv_mult << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >>> (padded_img_d, kernel_d, res + i * total_pixel_num,
	//		lrImage.dim_x, lrImage.dim_y, lrImage.dim_z, kernel_dim, total_pixel_num);
	//	/*hipMemcpy(res_c.data(), res + i * total_pixel_num, sizeof(float) * 128 * 128 * 128, hipMemcpyDeviceToHost);
	//	res_C.push_back(res_c);*/
	//}
	
	for (int i = 0; i < kernels.size(); ++i)
	{
		float* kernel_d;
		hipMalloc(&kernel_d, sizeof(float) * kernel_dim * kernel_dim * kernel_dim);
		hipMemcpy(kernel_d, kernels[i].data(), sizeof(float) * kernel_dim * kernel_dim * kernel_dim, hipMemcpyHostToDevice);

		Eigen::Tensor<float, 3> res_c(128, 128, 128);
		conv_mult << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (padded_img_d, kernel_d, res + i * total_pixel_num,
			lrImage.dim_x, lrImage.dim_y, lrImage.dim_z, kernel_dim, total_pixel_num);
		/*hipMemcpy(res_c.data(), res + i * total_pixel_num, sizeof(float) * 128 * 128 * 128, hipMemcpyDeviceToHost);
		res_C.push_back(res_c);*/
	}

	unsigned char *id_res;
	hipMalloc(&id_res, sizeof(unsigned char*) * total_pixel_num);
	obtain_ids << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (id_res, res, total_pixel_num, kernels.size() );
	hipMemcpy(id_res_c.data(), id_res, sizeof(unsigned char) * total_pixel_num, hipMemcpyDeviceToHost);

	// free memory
	{
		hipFree(lrImage_d);
		hipFree(padded_img_d);
		hipFree(res);
		hipFree(id_res);
	}
}


__global__ void initialize(float *src, float val, int num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num)
	{
		src[idx] = val;
	}
}


__global__ void initialize(unsigned short *src, unsigned short val, int num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num)
	{
		src[idx] = val;
	}
}

//__global__ void initialize(unsigned short *src, unsigned short val, int num)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	if (idx < num)
//	{
//		src[idx] = val;
//	}
//}


extern "C" void test_conv(std::vector<Eigen::Tensor<float, 3>> &oris, Eigen::Tensor<float, 3> &kernel, std::vector<Eigen::Tensor<float, 3>> &output)
{
	int padding = 1;
	int kernel_dim = 3;
	for (int ori_id = 0; ori_id < oris.size(); ori_id++)
	{
		float* lrImage_d;
		hipMalloc(&lrImage_d, sizeof(float) * oris[ori_id].dimension(0) * oris[ori_id].dimension(1) * oris[ori_id].dimension(2));
		hipMemcpy(lrImage_d, oris[ori_id].data(), sizeof(float) * oris[ori_id].dimension(0) * oris[ori_id].dimension(1) * oris[ori_id].dimension(2),
			hipMemcpyHostToDevice);
		
		float* padded_img_d;
		hipMalloc(&padded_img_d, sizeof(float) * (oris[ori_id].dimension(0) + padding * 2) * (oris[ori_id].dimension(1) + padding * 2)
			* (oris[ori_id].dimension(2) + padding * 2));
	/*	hipMemset(padded_img_d, 1.0, sizeof(float) * (oris[ori_id].dimension(0) + padding * 2) * (oris[ori_id].dimension(1) + padding * 2)
			* (oris[ori_id].dimension(2) + padding * 2));*/
		initialize << < ((oris[ori_id].dimension(0) + padding * 2) * (oris[ori_id].dimension(1) + padding * 2)
			* (oris[ori_id].dimension(2) + padding * 2) + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (padded_img_d, 0.0f,
			(oris[ori_id].dimension(0) + padding * 2) * (oris[ori_id].dimension(1) + padding * 2)
				* (oris[ori_id].dimension(2) + padding * 2));

		int total_pixel_num = oris[ori_id].dimension(0) * oris[ori_id].dimension(1) * oris[ori_id].dimension(2);
		pad_img << <(total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (lrImage_d,
			padded_img_d, padding, padding, padding, total_pixel_num, oris[ori_id].dimension(0), oris[ori_id].dimension(1), oris[ori_id].dimension(2));
		Eigen::Tensor<float, 3> padded_img(oris[ori_id].dimension(0) + 2, oris[ori_id].dimension(1) + 2, oris[ori_id].dimension(2) + 2);
		hipMemcpy(padded_img.data(), padded_img_d, sizeof(float) * (oris[ori_id].dimension(0) + padding * 2) * (oris[ori_id].dimension(1) + padding * 2) * (oris[ori_id].dimension(2) + padding * 2), hipMemcpyDeviceToHost);
		
		float* kernel_d;
		hipMalloc(&kernel_d, sizeof(float) * kernel_dim * kernel_dim * kernel_dim);
		hipMemcpy(kernel_d, kernel.data(), sizeof(float) * kernel_dim * kernel_dim * kernel_dim, hipMemcpyHostToDevice);

		float *res;
		hipMalloc(&res, sizeof(float) * oris[ori_id].dimension(0) * oris[ori_id].dimension(1) * oris[ori_id].dimension(2));
		conv_mult << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (padded_img_d, kernel_d, res, oris[ori_id].dimension(0),
			oris[ori_id].dimension(1), oris[ori_id].dimension(2), 3, total_pixel_num);

		Eigen::Tensor<float, 3> tmp_res(oris[ori_id].dimension(0), oris[ori_id].dimension(1), oris[ori_id].dimension(2));
		hipMemcpy(tmp_res.data(), res, sizeof(float) * oris[ori_id].dimension(0) * oris[ori_id].dimension(1) * oris[ori_id].dimension(2), hipMemcpyDeviceToHost);
		output.push_back(tmp_res);
	}
}


extern "C" void calc_direction_id(std::vector<Eigen::Tensor<float, 3> > kernels, Voxel_L &lrImage, std::vector<Eigen::Tensor<float, 3> > &res,
	Eigen::Tensor<unsigned char, 3> &idx_res)
{
	conv_new(lrImage, kernels, lrImage.dim_x, lrImage.dim_y, lrImage.dim_z, kernels[0].dimension(0), true, int((kernels[0].dimension(0) - 1) / 2), res,
		idx_res);
}


void calc_mean_std(unsigned short *lrImage_d, int total_pixel_num, double mean_res, double std_res)
{
	
}



extern "C" void fast_iter2(int level, Voxel_L &lrImage, Eigen::Tensor<unsigned char, 3> &dirSet, Eigen::Tensor<float, 3> &ss,
	int ss_x, int ss_y, int ss_z, Voxel_E &predict, int type_id, double thred, double thred2)
{
	unsigned short* lrImage_d;
	hipMalloc(&lrImage_d, sizeof(unsigned short) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z);
	for (int i = 0; i < lrImage.dim_z; ++i)
	{
		hipMemcpy(lrImage_d + i * lrImage.dim_x * lrImage.dim_y, lrImage.data_ptr[i], sizeof(unsigned short) *
			lrImage.dim_x * lrImage.dim_y, hipMemcpyHostToDevice);
	}

	unsigned char* dirset_d;
	hipMalloc(&dirset_d, sizeof(unsigned char) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z);
	hipMemcpy(dirset_d, dirSet.data(), sizeof(unsigned char) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z, hipMemcpyHostToDevice);
	
	float *ss_d;
	hipMalloc(&ss_d, sizeof(float) * ss_x * ss_y * ss_z);
	hipMemcpy(ss_d, ss.data(), sizeof(float) * ss_x * ss_y * ss_z, hipMemcpyHostToDevice);

	const int d = lrImage.dim_z, w = lrImage.dim_y, h = lrImage.dim_x;
	unsigned char *Predict;
	hipMalloc(&Predict, sizeof(unsigned char) * w * h * d);
	int total_pixel_num = d * w * h;

	double *mean_d, mean = 0.0;
	int *valid_num, valid = 0;
	hipMalloc(&valid_num, sizeof(int));
	hipMemcpy(valid_num, &valid, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&mean_d, sizeof(double));
	hipMemcpy(mean_d, &mean, sizeof(double), hipMemcpyHostToDevice);
	double std = 0.0;
	double *std_d;
	hipMalloc(&std_d, sizeof(double));
	hipMemcpy(std_d, &std, sizeof(double), hipMemcpyHostToDevice);
	
	calc_valid_number << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (lrImage_d, total_pixel_num, valid_num);
	hipMemcpy(&valid, valid_num, sizeof(int), hipMemcpyDeviceToHost);

	calc_mean << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (lrImage_d, total_pixel_num, mean_d, valid);
	hipMemcpy(&mean, mean_d, sizeof(double), hipMemcpyDeviceToHost);
	calc_std << < (total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (lrImage_d, total_pixel_num, mean, std_d, valid);
	hipMemcpy(&std, std_d, sizeof(double), hipMemcpyDeviceToHost);
	//std = 40.0 * 40.0;
	if(level == 0)
		printf("Valid: %d, Mean : %f, Std : %f\n", valid, mean, std::sqrt(std));

	calc_feature_map_with_meanstd << <(total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (lrImage_d, dirset_d, ss_d, h, w, d, Predict,
		total_pixel_num, ss_x, ss_y, ss_z, level, type_id, thred, thred2, mean, std::sqrt(std) );

	for (int i = 0; i < predict.dim_z; ++i)
	{
		hipMemcpy(predict.data_ptr[i], Predict + i * predict.dim_x * predict.dim_y, sizeof(unsigned char) * predict.dim_x * predict.dim_y, hipMemcpyDeviceToHost);
	}
	hipFree(lrImage_d);
	hipFree(dirset_d);
	hipFree(ss_d);
	hipFree(Predict);
	hipFree(valid_num);
	hipFree(mean_d);
	hipFree(std_d);
}


extern "C" void fast_iter(int level, Voxel_L &lrImage, Voxel_E &dirSet, Eigen::Tensor<float, 3> &ss,
	int ss_x, int ss_y, int ss_z, Voxel_E &predict, int type_id)
{
	unsigned short* lrImage_d;
	hipMalloc(&lrImage_d, sizeof(unsigned short) * lrImage.dim_x * lrImage.dim_y * lrImage.dim_z);
	for (int i = 0; i < lrImage.dim_z; ++i)
	{
		hipMemcpy(lrImage_d + i * lrImage.dim_x * lrImage.dim_y, lrImage.data_ptr[i], sizeof(unsigned short) *
			lrImage.dim_x * lrImage.dim_y, hipMemcpyHostToDevice);
	}

	unsigned char* dirset_d;
	hipMalloc(&dirset_d, sizeof(unsigned char) * dirSet.dim_x * dirSet.dim_y * dirSet.dim_z);
	for (int i = 0; i < dirSet.dim_z; ++i)
	{
		hipMemcpy(dirset_d + i * dirSet.dim_x * dirSet.dim_y, dirSet.data_ptr[i], sizeof(unsigned char) * dirSet.dim_x * dirSet.dim_y, hipMemcpyHostToDevice);
	}

	float *ss_d;
	hipMalloc(&ss_d, sizeof(float) * ss_x * ss_y * ss_z);
	hipMemcpy(ss_d, ss.data(), sizeof(float) * ss_x * ss_y * ss_z, hipMemcpyHostToDevice);

	const int d = lrImage.dim_z, w = lrImage.dim_y, h = lrImage.dim_x;
	unsigned char *Predict;
	hipMalloc(&Predict, sizeof(unsigned char) * w * h * d);
	int total_pixel_num = d * w * h;
	calc_feature_map<<<(total_pixel_num + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(lrImage_d, dirset_d, ss_d, h, w, d, Predict,
		total_pixel_num, ss_x, ss_y, ss_z, level, type_id, 20.0, 1.0);
	

	for (int i = 0; i < predict.dim_z; ++i)
	{
		hipMemcpy(predict.data_ptr[i], Predict + i * predict.dim_x * predict.dim_y, sizeof(unsigned char) * predict.dim_x * predict.dim_y, hipMemcpyDeviceToHost);
	}
	hipFree(lrImage_d);
	hipFree(dirset_d);
	hipFree(ss_d);
	hipFree(Predict);
}